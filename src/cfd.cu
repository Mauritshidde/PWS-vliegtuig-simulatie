#include "hip/hip_runtime.h"
#include "cfd.h"
#include <hip/hip_runtime.h>

#define WITHOUT_NUMPY
#include "Physics/matplotlibcpp.h"

namespace mat = matplotlibcpp;

void Cfd::Start() {
    
}

void Cfd::createMesh()
{
    for (int i = 0; i < nz; i++)
    {   
        std::vector<std::vector<MeshCube>> helper;
        std::vector<std::vector<double>> helper2;
        std::vector<std::vector<Vector3>> helper3;
        for (int j = 0; j < nx; j++)
        {
            std::vector<MeshCube> helperHelper;
            std::vector<double> helperHelper2;
            std::vector<Vector3> helperHelper3;
            for (int k = 0; k < ny; k++)
            {
                helperHelper.push_back(MeshCube());
                helperHelper2.push_back(0);
                helperHelper3.push_back({0,0,0});
            }
            helper.push_back(helperHelper);
            helper2.push_back(helperHelper2);
            helper3.push_back(helperHelper3);
        }
        mesh.push_back(helper);
        divergenceVelocityScalarField.push_back(helper2);
        gradientPressureField.push_back(helper3);
        divergenceVelocityField.push_back(helper3);
        divergenceFreeField.push_back(helper3);
        tempVelocity.push_back(helper3);
    }
}

void Cfd::setBoundaryConditions(double velocityXDirectionStart, double velocityYDirectionStart, double velocityZDirectionStart, double velocityXDirectionEnd, double velocityYDirectionEnd, double velocityZDirectionEnd)
{
    for (int i = 0; i < nz; i++)
    {
        for (int k = 0; k < ny; k++)
        {
            mesh.at(i).at(0).at(k).boundary = true;
            mesh.at(i).at(0).at(k).velocityX = velocityXDirectionStart;
            // mesh.at(i).at(0).at(k).pressure = pow(velocityXDirectionStart,2) * (rho/2.0f); // quess for starting pressure

            mesh.at(i).at(nx - 1).at(k).boundary = true;
            mesh.at(i).at(nx - 1).at(k).velocityX = velocityXDirectionEnd;
            // mesh.at(i).at(nx - 1).at(k).pressure = ; // set the pressure of the boundary
        }
    }

    for (int i = 0; i < nz; i++)
    {
        for (int j = 0; j < nx; j++)
        {
            mesh.at(i).at(j).at(0).boundary = true;
            mesh.at(i).at(j).at(0).velocityY = velocityYDirectionStart;

            mesh.at(i).at(j).at(ny - 1).boundary = true;
            mesh.at(i).at(j).at(ny - 1).velocityY = velocityYDirectionEnd;
        }
    }

    for (int j = 0; j < nx; j++)
    {
        for (int k = 0; k < ny; k++)
        {
            mesh.at(0).at(j).at(k).boundary = true;
            mesh.at(0).at(j).at(k).velocityZ = velocityZDirectionStart;

            mesh.at(nz - 1).at(j).at(k).boundary = true;
            mesh.at(nz - 1).at(j).at(k).velocityZ = velocityZDirectionEnd;
        }
    }
}

__device__ bool getCollisionPlaneRay(Vector3 direction, Vector3 oppositeDirection, Ray ray, Ray ray2) {
    ray.direction = direction;
    ray2.direction = oppositeDirection;

    // RayCollision meshHitInfo = GetRayCollisionMesh(ray, *airplane.meshes, airplane.transform);
    // RayCollision meshHitInfo2 = GetRayCollisionMesh(ray2, *airplane.meshes, airplane.transform);
    
    // if (meshHitInfo.hit && meshHitInfo2.hit) {
    //     return true;
    // } else {
    //     return false;
    // }
}

__global__ void setPlaneBoundaryHelper(MeshCube *mesh, Vector3 startingPoint, int dx, int dy, int dz, int i, int j) {
    Vector3 position;
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    position.x = dx * j + startingPoint.x;
    position.y = dy * k + startingPoint.y;
    position.z = dz * i + startingPoint.z;
    // if (CheckCollisionBoxSphere(boundingBoxPlane, position, dx)) {
        Ray ray;
        Ray ray2;
        
        ray.position = position;
        ray2.position = position;
        
        // check if the cube is inside the plane
        if (getCollisionPlaneRay({1,0,0}, {1,0,0}, ray, ray2)) { 
            if (getCollisionPlaneRay({0,1,0}, {0,1,0}, ray, ray2)) {
                if (getCollisionPlaneRay({0,0,1}, {0,0,1}, ray, ray2)) {
                    mesh[k].boundary = true;
                }
            }
        }
    // }
}

void Cfd::setPlaneBoundary() //222
{
    int N2 = mesh.at(0).at(0).size();
    // int threadsPerBlock = 256;
    // int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    size_t size = sizeof(MeshCube) * N2;
    for (int i=1; i < nz-1; i++) {
        for (int j=1; j < nx-1; j++) {
        
            MeshCube *array = mesh.at(i).at(j).data();
            MeshCube *array_p;

            hipMalloc(&array_p, size);
            hipMemcpy(array_p, array, size, hipMemcpyHostToDevice);
            
            setPlaneBoundaryHelper<<<grid_size, block_size>>>(array_p, startingPoint, dx, dy, dz, i, j);

            hipMemcpy(array, array_p, size, hipMemcpyDeviceToHost);
        }
    }
    std::cout << "done this" << std::endl;
}

void Cfd::resetMesh() {
    for (int i = 1; i < nz-1; i++)
    {   
        for (int j = 1; j < nx-1; j++)
        {
            for (int k = 1; k < ny-1; k++)
            {
                mesh.at(i).at(j).at(k) = MeshCube();
                divergenceVelocityField.at(i).at(j).at(k) = {0,0,0};
            }
        }
    }
}

__global__ void velocityMovementHelper(int N, int M, int B, MeshCube *mesh, float dT, double rho, double dx, double dy, double dz) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    int k = index / (M * B);
    int j = (index % (M * B)) / B;
    int i = (index % B);

    if (i < N * M * B && (i > 0 && i < N-1) && (j > 0 && j < M-1) && (k > 0 && k < B-1)) {
        if (!mesh[index].boundary) {
            // int index = z * (nx * ny) + x * ny + y;
            double vx = mesh[index].velocityX;
            double vy = mesh[index].velocityY;
            double vz = mesh[index].velocityZ;
            double duDt = -(vx * (vx - mesh[k + i*M * B + (j-1) * B].velocityX) / dx +
                    vy * (vx - mesh[(k-1) + i*M * B + j * B].velocityX) / dy + 
                    vz * (vx - mesh[k + (i-1)*M * B + j * B].velocityX) / dz) / dx;

            double dvDt = -(vx * (vy - mesh[k + i*M * B + (j-1) * B].velocityY) / dx +
                    vy * (vy - mesh[(k-1) + i*M * B + j * B].velocityY) / dy + 
                    vz * (vy - mesh[k + (i-1)*M * B + j * B].velocityY) / dz) / dy;

            double dwDt = -(vx * (vz - mesh[k + i*M * B + (j-1) * B].velocityZ) / dx +
                    vy * (vz - mesh[(k-1) + i*M * B + j * B].velocityZ) / dy +
                    vz * (vz - mesh[k + (i-1)*M * B + j * B].velocityZ) / dz) / dz;

            mesh[index].tempVelocity.x = mesh[index].velocityX + duDt * dT;
            mesh[index].tempVelocity.y = mesh[index].velocityY + dvDt * dT;
            mesh[index].tempVelocity.z = mesh[index].velocityZ + dwDt * dT;
            mesh[index].pressure = (duDt / dT + dvDt / dT + dwDt / dT) * rho; 
        }
    }
}

__global__ void velocityMovementUpdater(int N, int M, int B, MeshCube *mesh) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    int k = index / (M * B);
    int j = (index % (M * B)) / B;
    int i = (index % B);

    if (i < N * M * B && (i > 0 && i < N-1) && (j > 0 && j < M-1) && (k > 0 && k < B-1)) {
        // int index = z * (nx * ny) + x * ny + y;
        if (!mesh[index].boundary) {
            mesh[k + i*M * B + j * B].velocityX = mesh[k + i*M * B + j * B].tempVelocity.x;
            mesh[k + i*M * B + j * B].velocityX = mesh[k + i*M * B + j * B].tempVelocity.y;
            mesh[k + i*M * B + j * B].velocityX = mesh[k + i*M * B + j * B].tempVelocity.z; 
        }
    }
}

// void Cfd::velocityMovement(float dT) {
//     // std::cout << "start " << std::endl;
//     // int N = mesh.at(0).at(0).size();
//     // int M = mesh.at(0).size();
//     // int B = mesh.size();
    

//     // auto start = std::chrono::system_clock::now();
//     velocityMovementHelper<<<grid_size, block_size>>>(N, M, B, array_p, dT, rho, dx, dy, dz);
//     // auto end = std::chrono::system_clock::now();
//     // std::chrono::duration<double> elapsed_seconds = end-start;
//     // std::cout << elapsed_seconds.count() << std::endl;
// }

Vector3 Cfd::getNetPressureOnPlane() {
    Vector3 netPressure = {0,0,0};

    for (int i=1; i < nz-1; i++) {
        for (int j=1; j < nx-1; j++) {
            for (int k=1; k < ny-1; k++) {
                if (mesh.at(1).at(j).at(k).boundary) {
                    if (!mesh.at(i).at(j+1).at(k).boundary) {
                        netPressure.x += mesh.at(1).at(j+1).at(k).pressure * dy * dz;
                    }
                    if (!mesh.at(i).at(j-1).at(k).boundary) {
                        netPressure.x -= mesh.at(1).at(j-1).at(k).pressure * dy * dz;
                    }
                    if (!mesh.at(i).at(j).at(k+1).boundary) {
                        netPressure.y += mesh.at(1).at(j).at(k+1).pressure * dx * dz;
                    }
                    if (!mesh.at(i).at(j).at(k-1).boundary) {
                        netPressure.y -= mesh.at(1).at(j).at(k-1).pressure * dx * dz;
                    }
                    if (!mesh.at(i+1).at(j).at(k).boundary) {
                        netPressure.z += mesh.at(i+1).at(j).at(k).pressure * dx * dy;
                    }
                    if (!mesh.at(i-1).at(j).at(k).boundary) {
                        netPressure.z -= mesh.at(i-1).at(j).at(k).pressure * dx * dy;
                    }
                }
            }
        }
    }

    return netPressure;
}

Vector2 Cfd::calc(double anglePitch, double angleYaw)
{
    float cl, cd;
    double tijd = 0;
    std::vector<std::vector<std::vector<double>>> *diffuseV;
    
    for (int i=0; i < N; i++) {
        
        for (int j=0; j < M; j++) {
            for (int k=0; k < B; k++) {
                mesh_array[k + i*M * B + j * B] = mesh.at(i).at(j).at(k);
            }
        }
    } 

    std::cout << "start loop for getting pressure and velocity" << std::endl;
    // setting gpu memory
    MeshCube *array_p;

    hipMalloc(&array_p, N * M * B * sizeof(MeshCube));
    hipMemcpy(array_p, mesh_array, N * M * B * sizeof(MeshCube), hipMemcpyHostToDevice);
    
    while (tijd < maxTime)
    {
        auto start = std::chrono::system_clock::now();
        tijd += dT;
        
        // TODO the movement of the pressure NOTE density is constant

        std::vector<std::thread> threads;
        int newNz = nz - 2;

        // std::cout << "start movement" << std::endl;
        // velocityMovement(dT);

        velocityMovementHelper<<<grid_size, block_size>>>(N, M, B, array_p, dT, rho, dx, dy, dz);
        velocityMovementUpdater<<<grid_size, block_size>>>(N, M, B, array_p);
        


        // std::cout << "end movement" << std::endl;

        // for (int i=1; i < nz-1; i++) {
        //     for (int j=1; j < nx-1; j++) {
        //         for (int k=1; k < ny-1; k++) {
        //             if (!mesh_array[k + i*M * B + j * B].boundary) {
        //                 mesh_array[k + i*M * B + j * B].velocityX = mesh_array[k + i*M * B + j * B].tempVelocity.x;
        //                 mesh_array[k + i*M * B + j * B].velocityX = mesh_array[k + i*M * B + j * B].tempVelocity.y;
        //                 mesh_array[k + i*M * B + j * B].velocityX = mesh_array[k + i*M * B + j * B].tempVelocity.z;
        //             }
        //         }
        //     }
        // }

        if (drawing) {
            // for (int i=0; i < N; i++) {
            //     for (int j=0; j < M; j++) {
            //         for (int k=0; k < B; k++) {
            //             mesh.at(i).at(j).at(k) = mesh_array[k + i*M * B + j * B];
            //         }
            //     }
            // } 
            Draw();
        }
        
        auto end = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::cout << elapsed_seconds.count() << std::endl;
        std::cout << tijd << std::endl;
    }

    // deleting array_p and writing gpu memory to cpu
    hipMemcpy(mesh_array, array_p, N * M * B * sizeof(MeshCube), hipMemcpyDeviceToHost);
    hipFree(array_p);

    std::cout << "done with loop getting pressure and velocity" << maxTime << std::endl;
    for (int i=0; i < N; i++) {
        for (int j=0; j < M; j++) {
            for (int k=0; k < B; k++) {
                mesh.at(i).at(j).at(k) = mesh_array[k + i*M * B + j * B];
            }
        }
    } 
    // TODO correction fase
    // correction 

    Vector3 forces = getNetPressureOnPlane();
    // TODO the 100 is the starting velocity of the boudnary on the left
    cl = forces.y / (rho * pow(10 ,2) * 0.5);
    cd = forces.x / (rho * pow(10 ,2) * 0.5);
    // float cz = forces.z / (rho * pow(100 ,2) * 0.5);

    return {cl, cd};
}

void Cfd::moveCamera(float deltaTime) {
    Vector2 currentMousePos = GetMousePosition();

    if (IsMouseButtonDown(0))
    {
        angleYAxis += 100 * ((currentMousePos.x - previousMousePosition.x)) * deltaTime;
        angleXZAxis += 100 * ((currentMousePos.y - previousMousePosition.y)) * deltaTime;
        if (angleYAxis > 360) {
            angleYAxis -= 360;
        } else if (angleYAxis < 0) {
            angleYAxis += 360;
        }

        if (angleXZAxis > 360) {
            angleYAxis -= 360;
        } else if (angleYAxis < 0) {
            angleXZAxis += 360;
        }
    }

    if (IsKeyDown(KEY_RIGHT))
    {
        angleYAxis += 100 * deltaTime;
        if (angleYAxis > 360) {
            angleYAxis -= 360;
        }
    }
    if (IsKeyDown(KEY_LEFT))
    {
        angleYAxis -= 100 * deltaTime;
        if (angleYAxis < 0) {
            angleYAxis += 360;
        }
    }
    if (IsKeyDown(KEY_UP))
    {
        angleXZAxis += 100 * deltaTime;
        if (angleXZAxis > 360) {
            angleXZAxis -= 360;
        }
    }
    if (IsKeyDown(KEY_DOWN))
    {
        angleXZAxis -= 100 * deltaTime;
        if (angleXZAxis < 0) {
            angleXZAxis += 360;
        }
    }

    if (GetMouseWheelMove() > 0)
    {
        cameraCircleRadius += 100 * deltaTime;
        cameraPos = {0.0f, 0.0f, cameraCircleRadius};
    }
    else if (GetMouseWheelMove() < 0)
    {
        cameraCircleRadius -= 100 * deltaTime;
        cameraPos = {0.0f, 0.0f, cameraCircleRadius};
    }

    camera.position = Vector3Transform2(cameraPos, MatrixRotateXYZ2((Vector3){DEG2RAD * angleXZAxis, DEG2RAD * angleYAxis, 0}));
    previousMousePosition = currentMousePos;
}

void Cfd::drawVelocityVectors() {
    for (int i=1; i < nz-1; i+=(nz/30)) {
        for (int j=1; j < nx-1; j+=(nx/(nx/2))) {
            for (int k=1; k < ny-1; k+=(ny/(ny/4))) {
                Vector3 point;
                point.x = startingPoint.x + j * dx - 0.5 * dx;
                point.y = startingPoint.y + k * dy - 0.5 * dy;
                point.z = startingPoint.z + i * dz - 0.5 * dz;
                if (mesh.at(i).at(j).at(k).boundary) {
                    // DrawCubeWires(point, dx, dy, dz, BLACK);
                    DrawCube(point, dx, dy, dz, BLACK);
                } else {
                    
                    float velocityX = mesh.at(i).at(j).at(k).velocityX;
                    float velocityY = mesh.at(i).at(j).at(k).velocityY;
                    float velocityZ = mesh.at(i).at(j).at(k).velocityZ;
                    float velocity = sqrt(pow(velocityX,2) + pow(velocityY,2) + pow(velocityZ,2));
                    
                    double val = (velocity / 200.0f) *300;
                    double val2 = (velocity / 500.0f);
                    double val3 = velocity * 180;
                    
                    Color velocityColor = {val, val2, val3, 255};
                    
                    Vector3 velocityDirection = {velocityX,velocityY,velocityZ};
                    velocityDirection = Vector3Normalize2(velocityDirection);
                    velocityDirection.x = (velocityDirection.x * 0.5 * dx + point.x);
                    velocityDirection.y = (velocityDirection.y * 0.5 * dy + point.y);
                    velocityDirection.z = (velocityDirection.z * 0.5 * dz + point.z);
                    // std::cout << point.x << "  x " << velocityDirection.x << std::endl;
                    // std::cout << point.y << " y " << velocityDirection.y << std::endl;
                    // std::cout << point.z << " z " << velocityDirection.z << std::endl;
                    DrawLine3D(point, velocityDirection, velocityColor); //111
                    // DrawLine3D(point, {point.x, point.y, point.z+dz}, BLUE);
                    // DrawCubeWires(point, dx, dy, dz, RED);
                    // std::cout << velocity << " ";
                }
            }
            // std::cout  << std::endl;
            // std::cout << "velocity start " << mesh.at(1).at(0).at(0).velocityX << std::endl; 
        }
    }
            // std::cout  << std::endl;
            // std::cout  << std::endl;
            // std::cout  << std::endl;
}

void Cfd::draw2DGrid() {
    for (int j=0; j < nx-1; j++) {
        for (int k=1; k < ny-1; k++) {
            Vector3 point;
            point.x = j * dx - 0.5 * dx;
            point.y = k * dy - 0.5 * dy;
            point.z = startingPoint.z + dz - 0.5 * dz;

            if (mesh_array[k + 1*M * B + j * B].boundary) {
                DrawRectangle(point.x*4, point.y*4, dx*4, dy*4, BLACK);

            } else {
                float velocityX = mesh_array[k + 1*M * B + j * B].velocityX;
                float velocityY = mesh_array[k + 1*M * B + j * B].velocityY;
                float velocityZ = mesh_array[k + 1*M * B + j * B].velocityZ;
                float velocity = sqrt(pow(velocityX,2) + pow(velocityY,2) + pow(velocityZ,2));
                
                double val = (velocity / 200.0f) *300;
                double val2 = (velocity / 500.0f);
                double val3 = velocity * 180;
                
                Color velocityColor = {255, val2, val3, 255};
                DrawRectangle(point.x*4, point.y*4, dx*4, dy*4, velocityColor);
                // std::cout << mesh.at(1).at(j).at(k).pressure << " ";
                // std::cout << velocity << " ";
            }
        }
        // std::cout  << std::endl;
    }
    // std::cout  << std::endl;
    // std::cout  << std::endl;
    // std::cout  << std::endl;
}

void Cfd::Draw() {
    moveCamera(GetFrameTime());
    BeginDrawing();
        ClearBackground(WHITE);
        if (drawing3D) {
            BeginMode3D(camera);
                drawVelocityVectors();
            EndMode3D();
        } else {
            draw2DGrid();
        }
    EndDrawing();
}

void Cfd::run(int steps, double stepsizePitch, double stepsizeYaw) { //333
    N = mesh.at(0).at(0).size();
    M = mesh.at(0).size();
    B = mesh.size();
    
    grid_size = ((N * M * B + 255) / block_size);
    mesh_array = (MeshCube*)malloc(N * M * B * sizeof(MeshCube));

    double stepsize = 360.0f/steps;
    std::vector<std::vector<Vector2>> cfdResults;
    for (double i=0; i <= 360; i+=stepsize) { // pitch
        std::vector<Vector2> cfdResultsHelper;
        for (double j=0; j <= 360; j+=stepsize) { // yaw
            airplane.transform = MatrixRotateXYZ2((Vector3){DEG2RAD * i, DEG2RAD * j, DEG2RAD * 0});
            resetMesh();
            // setPlaneBoundary();
                // for (int j=1; j < ny-1; j++) {
                //     mesh.at(1).at(nx/2).at(j).boundary = true;
                // }
                //     mesh.at(1).at(nx/2).at(ny/2-1).boundary = false;
                //     mesh.at(1).at(nx/2).at(ny/2-2).boundary = false;
                //     mesh.at(1).at(nx/2).at(ny/2+1).boundary = false;
                //     mesh.at(1).at(nx/2).at(ny/2+2).boundary = false;

            Vector2 consts = calc(i, j);
            cfdResultsHelper.push_back(consts);
        }
    }
    std::cout << "done" << std::endl;
    std::vector<Vector2> cfdResultsPitch, cfdResultsYaw;
    for (double i=0; i <= 360; i+=stepsizePitch) { // pitch
        airplane.transform = MatrixRotateXYZ2((Vector3){DEG2RAD * i, DEG2RAD * 0, DEG2RAD * 0});
        resetMesh();
        setPlaneBoundary();
        Vector2 consts = calc(i, 0);
        cfdResultsPitch.push_back({consts.x, consts.y});
    }
    std::cout << "done2" << std::endl;

    for (double i=0; i <= 360; i+=stepsizeYaw) {
        airplane.transform = MatrixRotateXYZ2((Vector3){DEG2RAD * 0, DEG2RAD * i, DEG2RAD * 0});
        resetMesh();
        setPlaneBoundary();

        Vector2 consts = calc(0, i);
        cfdResultsYaw.push_back({consts.x, consts.y});
    }
    std::cout << "done3" << std::endl;

    createLiftFiles(&cfdResults, &cfdResultsPitch, &cfdResultsYaw);
    if (drawing) {
        CloseWindow();
    }
    
    free(mesh_array);
    std::cout << "cfd-program completed calculating cl and cd over pitch and yaw and exited succesfully";
}

Cfd::Cfd(int setnx, int setny, int setnz, double deltaTime, double setMaxTime, double setRho, bool drawingEnabled, bool draw3D)
{   
    Re = 100;
    nu = 1 / Re;

    // set variables for gpu
    block_size = 2048;

    // set multithreading variables
    cores = 12;
    settingPlaneBOundarys = false;

    // set camera variables
    cameraCircleRadius = 150;
    cameraPos = {0.0f, 0.0f, cameraCircleRadius};
    cameraXYPos = {cameraPos.x, cameraPos.y};
    camera = {0};
    angleYAxis = 0;
    angleXZAxis = 0;

    camera.position = cameraPos;                  // Camera position perspective
    camera.target = (Vector3){0.0f, 0.0f, 0.0f}; // Camera looking at point  20 ?????????????? hier naar nog kijken TODO
    camera.up = (Vector3){0.0f, 30.0f, 0.0f};     // Camera up vector (rotation towards target)
    camera.fovy = 30.0f;                          // Camera field-of-view Y   effect van dit veranderen bestuderen ?????????????? TODO
    camera.projection = CAMERA_PERSPECTIVE;  /// wat doet dit TODO

    // set plane model variables
    airplane = LoadModel("models/object/airplane.obj");
    airplaneTexture = LoadTexture("models/texture/planeTextureBeter.png");
    airplane.materials[0].maps[MATERIAL_MAP_DIFFUSE].texture = airplaneTexture;
    airplane.transform = MatrixTranslate2(0, 0.0f, 0);

    // set simulation variables
    plane.loadObjectModel();

    boundingBoxPlane = GetModelBoundingBox(airplane);
    Vector3 boundingBoxPlaneMin = boundingBoxPlane.min;
    Vector3 boundingBoxPlaneMax = boundingBoxPlane.max;

    nx = setnx;
    ny = setny;
    nz = setnz;
    dT = deltaTime;
    maxTime = setMaxTime;
    rho = setRho;
    dx = 2;
    dy = 2;
    dz = 5;
    startingPoint.x = -(nx*dx)/2;
    startingPoint.y = -(ny*dy)/2 + 10;
    startingPoint.z = -(nz*dz)/2;
    
    drawing3D = draw3D;
    drawing = drawingEnabled;
    if (!drawingEnabled) {
        CloseWindow();
    }

    // functions for generating the mesh
    createMesh();
    setBoundaryConditions(10,  0,  0,  0,  0,  0);
}
 
Cfd::~Cfd()
{
}
