
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addvec(int N, int M, int B, double *array, double *sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int j = blockIdx.y * blockDim.y + threadIdx.y;
    // int index = j + i * M;
    if (i < N * M * B) {
        // sum[0] += array[i];
        __syncthreads();
        array[i] = 3;
        // printf("%d ",i);
    } else {
        __syncthreads();
        sum[0] = i;
    }
    
}

int main() {
    int N = 40;
    int M = 40;
    int B = 40;
    int block_size = 256;
    int grid_size = ((N * M * B + 255) / block_size);
    // dim3 threadsPerBlock((N+255)/256, (M+255)/256, (B+255)/256);

    double *array = (double*)malloc(N * M * B * sizeof(double));
    double *sum = (double*)malloc(sizeof(double));
    sum[0] = 2;

    for (int i=0; i < N; i++) {
        for (int j=0; j < M; j++) {
            for (int k=0; k < B; k++) {
                array[k + i*M + j * B] = 1;
            }
        }
    } 

    double *array_p;
    double *sum_p;

    hipMalloc(&array_p, N * M * B *sizeof(double));
    hipMemcpy(array_p, array, N * M * B * sizeof(double), hipMemcpyHostToDevice);
    
    hipMalloc(&sum_p, sizeof(double));
    hipMemcpy(sum_p, sum, sizeof(double), hipMemcpyHostToDevice);

    addvec<<<grid_size, block_size>>>(N, M, B, array_p, sum_p);

    hipMemcpy(array, array_p, N * M * B * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sum, sum_p, sizeof(double), hipMemcpyDeviceToHost);


    // for (int i=0; i < N * M; i++) {
    //     std::cout << array[i] << " ";
    // }
    for (int i=0; i < N; i++) {
        for (int j=0; j < M; j++) {
            for (int k=0; k < B; k++) {
                std::cout << array[j + i*M] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
        std::cout << std::endl;
    } 
    std::cout << std::endl;
    std::cout << std::endl;
    std::cout << sum[0] << " the sum " << std::endl;

    hipFree(array_p);
    free(array);
    
    hipFree(sum_p);
    free(sum);

    return 0;
}